#include <iostream>
#include "material.hh"

__host__
DeviceMaterial::DeviceMaterial(const host_mat_t& m)
{
    kd = m.kd;
    ka = m.ka;
    tex_w = m.tex_w;
    tex_h = m.tex_h;

    hipMalloc(&kd_map, sizeof(vec3_t) * m.kd_map.size());
    hipMemcpy(kd_map, m.kd_map.data(), m.kd_map.size() * sizeof(vec3_t), hipMemcpyHostToDevice);

    std::cout << "w*h = " << tex_w * tex_h << '\n'
              << "vec.size() = " << m.kd_map.size() << '\n';
}

__host__
DeviceMaterial::~DeviceMaterial()
{
    std::cout << "DTOR\n";
    hipFree(kd_map);
}

__host__
DeviceMaterial& DeviceMaterial::operator=(DeviceMaterial&& m)
{
    kd = m.kd;
    ka = m.kd;
    tex_w = m.tex_w;
    tex_h = m.tex_h;
    kd_map = m.kd_map;
    m.kd_map = nullptr;

    return *this;
}
