#include "material.hh"

__host__
DeviceMaterial::DeviceMaterial(const host_mat_t& m)
{
    kd = m.kd;
    ka = m.ka;
    tex_w = m.tex_w;
    tex_h = m.tex_h;

    hipMalloc(&kd_map, sizeof(vec3_t) * m.kd_map.size());
    hipMemcpy(kd_map, m.kd_map.data(), m.kd_map.size(), hipMemcpyHostToDevice);
}

__host__
DeviceMaterial::~DeviceMaterial()
{
    hipFree(kd_map);
}
